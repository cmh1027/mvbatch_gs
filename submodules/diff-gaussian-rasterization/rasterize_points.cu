#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include "cuda_rasterizer/utils.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		t.fill_(0);
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, float, float, float>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const torch::Tensor& tan_fovx, 
	const torch::Tensor& tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	torch::Tensor& mask,
	const int HS,
	const torch::Tensor& visibility_mapping, // (P, 1) => max values HS
	const bool write_visibility,
	const bool time_check,
	const bool debug)
{
	if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
		AT_ERROR("means3D must have dimensions (num_points, 3)");
	}
	auto PH = (image_height + BLOCK_Y - 1) / BLOCK_Y;
	auto PW = (image_width + BLOCK_X - 1) / BLOCK_X;
	if(mask.contiguous().data<int>() == nullptr){
		mask = torch::arange(BLOCK_X*BLOCK_Y, means3D.options().dtype(torch::kInt32)).unsqueeze(0).repeat({PH*PW, 1}); // (PH*PW, BLOCK_X * BLOCK_Y)
	}
	assert(mask.size(0) == PH*PW);
	assert(mask.size(1) == BLOCK_X * BLOCK_Y);
	assert(BLOCK_X == BLOCK_Y);

	const int P = means3D.size(0);
	const int H = image_height;
	const int W = image_width;
	const int B = viewmatrix.size(0);
	assert(BLOCK_X * BLOCK_Y % B == 0);

	auto int_opts = means3D.options().dtype(torch::kInt32);
	auto float_opts = means3D.options().dtype(torch::kFloat32);

	torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
	torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor out_trans = torch::full({1, H, W}, 0.0, float_opts);
	torch::Tensor radii = torch::full({B, P}, 0, means3D.options().dtype(torch::kInt32));
	
	torch::Device device(torch::kCUDA);
	torch::TensorOptions options(torch::kByte);
	torch::Tensor cacheBuffer = torch::empty({0}, options.device(device));
	torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
	torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
	torch::Tensor imgBuffer = torch::empty({0}, options.device(device));

	std::function<char*(size_t)> cacheFunc = resizeFunctional(cacheBuffer);
	std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
	std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
	std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

	torch::Tensor focal_y = H / (2.0f * tan_fovy);
	torch::Tensor focal_x = W / (2.0f * tan_fovx);

	torch::Tensor gaussian_visibility = torch::full({B, HS}, 0, means3D.options().dtype(torch::kInt32));

	int rendered = 0;
	int batch_rendered = 0;
	double measureTime, preprocessTime, renderTime;
	if(P != 0)
	{
		int M = 0;
		if(sh.size(0) != 0)
		{
			M = sh.size(1);
		}

		auto returned = CudaRasterizer::Rasterizer::forward(
			geomFunc,
			binningFunc,
			imgFunc,
			cacheFunc,
			P, degree, M, B, HS,
			background.contiguous().data<float>(),
			W, H,
			means3D.contiguous().data<float>(),
			sh.contiguous().data_ptr<float>(),
			opacity.contiguous().data<float>(), 
			scales.contiguous().data_ptr<float>(),
			scale_modifier,
			rotations.contiguous().data_ptr<float>(),
			viewmatrix.contiguous().data<float>(), 
			projmatrix.contiguous().data<float>(),
			campos.contiguous().data<float>(),
			focal_x.contiguous().data<float>(),
			focal_y.contiguous().data<float>(),
			tan_fovx.contiguous().data<float>(),
			tan_fovy.contiguous().data<float>(),
			out_color.contiguous().data<float>(),
			out_depth.contiguous().data<float>(),
			out_trans.contiguous().data<float>(),
			radii.contiguous().data<int>(),
			gaussian_visibility.contiguous().data<int>(),
			visibility_mapping.contiguous().data<int>(),
			write_visibility,
			mask.contiguous().data<int>(),
			time_check,
			debug
		);
		rendered = std::get<0>(returned);
		batch_rendered = std::get<1>(returned);
		measureTime = std::get<2>(returned);
		preprocessTime = std::get<3>(returned);
		renderTime = std::get<4>(returned);
	}
	return std::make_tuple(rendered, batch_rendered, out_color, out_depth, out_trans, radii, cacheBuffer, geomBuffer, binningBuffer, imgBuffer, gaussian_visibility, measureTime, preprocessTime, renderTime);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, float, float>
RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const torch::Tensor& tan_fovx,
	const torch::Tensor& tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& dL_dout_depth,
	const torch::Tensor& dL_dout_trans,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& cacheBuffer,
	const torch::Tensor& geomBuffer,
	const int R,
	const int BR,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const torch::Tensor& mask,
	const int grad_sep,
	const bool time_check,
	const bool debug)
{
	const int B = viewmatrix.size(0);
	const int P = means3D.size(0);
	const int H = dL_dout_color.size(1);
	const int W = dL_dout_color.size(2);

	torch::Tensor focal_y = H / (2.0f * tan_fovy);
	torch::Tensor focal_x = W / (2.0f * tan_fovx);

	int M = 0;
	if(sh.size(0) != 0)
	{	
		M = sh.size(1);
	}


	torch::Tensor dL_dmeans2D = torch::zeros({BR, 2}, means3D.options());
	torch::Tensor dL_dmeans2D_sq = torch::zeros({BR, 1}, means3D.options());
	
	torch::Tensor dL_dcolors = torch::zeros({BR, NUM_CHANNELS}, means3D.options());
	torch::Tensor dL_ddepths = torch::zeros({BR, 1}, means3D.options());
	torch::Tensor dL_dconic = torch::zeros({BR, 2, 2}, means3D.options());
	torch::Tensor dL_dcov3D = torch::zeros({BR, 6}, means3D.options());

	torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
	torch::Tensor point_idx = torch::zeros({BR, 1}, means3D.options().dtype(torch::kInt32));

	torch::Tensor dL_dmeans3D = torch::zeros({BR, 3}, means3D.options());
	torch::Tensor dL_dscales = torch::zeros({BR, 3}, means3D.options());
	torch::Tensor dL_drotations = torch::zeros({BR, 4}, means3D.options());
	torch::Tensor dL_dsh = torch::zeros({BR, M*3}, means3D.options());

	torch::Tensor dL_dmeans2D_sum = torch::zeros({P, 2}, means3D.options());
	torch::Tensor dL_dmeans3D_sum = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_dscales_sum = torch::zeros({P, 3}, means3D.options());
	torch::Tensor dL_drotations_sum = torch::zeros({P, 4}, means3D.options());
	torch::Tensor dL_dsh_sum = torch::zeros({P, M*3}, means3D.options());

	double preprocessTime, renderTime;
	if(BR != 0)
	{  
		auto returned = CudaRasterizer::Rasterizer::backward(P, degree, M, B, R, BR,
		background.contiguous().data<float>(),
		W, H, 
		means3D.contiguous().data<float>(),
		sh.contiguous().data<float>(),
		scales.data_ptr<float>(),
		scale_modifier,
		rotations.data_ptr<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		focal_x.contiguous().data<float>(),
		focal_y.contiguous().data<float>(),
		tan_fovx.contiguous().data<float>(),
		tan_fovy.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		reinterpret_cast<char*>(cacheBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
		dL_dout_color.contiguous().data<float>(),
		dL_dout_depth.contiguous().data<float>(),
		dL_dout_trans.contiguous().data<float>(),
		dL_dmeans2D.contiguous().data<float>(),
		dL_dmeans2D_sq.contiguous().data<float>(),
		dL_dconic.contiguous().data<float>(),  
		dL_dopacity.contiguous().data<float>(),
		dL_dcolors.contiguous().data<float>(),
		dL_ddepths.contiguous().data<float>(),
		dL_dmeans3D.contiguous().data<float>(),
		dL_dcov3D.contiguous().data<float>(),
		dL_dsh.contiguous().data<float>(),
		dL_dscales.contiguous().data<float>(),
		dL_drotations.contiguous().data<float>(),
		mask.contiguous().data<int>(),
		point_idx.contiguous().data<int>(),
		time_check,
		debug);
	
		preprocessTime = std::get<0>(returned);
		renderTime = std::get<1>(returned);

		point_idx = point_idx.to(torch::kInt64);

		dL_dmeans3D_sum.scatter_add_(0, point_idx.expand({-1, 3}), dL_dmeans3D);
		dL_dscales_sum.scatter_add_(0, point_idx.expand({-1, 3}), dL_dscales);
		dL_drotations_sum.scatter_add_(0, point_idx.expand({-1, 4}), dL_drotations);
		dL_dsh_sum.scatter_add_(0, point_idx.expand({-1, M*3}), dL_dsh);
		dL_dsh_sum = dL_dsh_sum.reshape({P, M, 3});

		torch::Tensor dL_dmeans2D_clone = torch::empty({0}, means3D.options());
		torch::Tensor dL_dmeans2D_split = torch::empty({0}, means3D.options());
		switch(grad_sep){
			case 0:
				dL_dmeans2D_clone = torch::zeros({P, 2}, means3D.options());
				dL_dmeans2D_split = torch::zeros({P, 2}, means3D.options());
				dL_dmeans2D_clone.scatter_add_(0, point_idx.expand({-1, 2}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}));
				dL_dmeans2D_split.scatter_add_(0, point_idx.expand({-1, 2}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}));
				dL_dmeans2D_sum = torch::cat({dL_dmeans2D_clone.norm(2, -1, true), dL_dmeans2D_split.norm(2, -1, true)}, -1);
				break;
			case 1:
				dL_dmeans2D_clone = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_split = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_clone.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}).norm(2, -1, true));
				dL_dmeans2D_split.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D_sq);
				dL_dmeans2D_sum = torch::cat({dL_dmeans2D_clone,dL_dmeans2D_split}, -1);
				break;
			case 2:
				dL_dmeans2D_clone = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_split = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_clone.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D_sq);
				dL_dmeans2D_split.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}).norm(2, -1, true));
				dL_dmeans2D_sum = torch::cat({dL_dmeans2D_clone,dL_dmeans2D_split}, -1);
				break;
			case 3:
				dL_dmeans2D_clone = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_split = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_clone.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D_sq);
				dL_dmeans2D_split.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D_sq);
				dL_dmeans2D_sum = torch::cat({dL_dmeans2D_clone,dL_dmeans2D_split}, -1);
				break;
			case 4:
				dL_dmeans2D_clone = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_split = torch::zeros({P, 1}, means3D.options());
				dL_dmeans2D_clone.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}).norm(2, -1, true));
				dL_dmeans2D_split.scatter_add_(0, point_idx.expand({-1, 1}), dL_dmeans2D.index({torch::indexing::Slice(), torch::indexing::Slice(0, 2)}).norm(2, -1, true));
				dL_dmeans2D_sum = torch::cat({dL_dmeans2D_clone,dL_dmeans2D_split}, -1);
				break;
			default:
				printf("Invalid gradient separation\n");
				exit(0);
		}
	}
	ERROR_CHECK
  	return std::make_tuple(dL_dmeans2D_sum, dL_dopacity, dL_dmeans3D_sum, dL_dsh_sum, dL_dscales_sum, dL_drotations_sum, preprocessTime, renderTime);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix
)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}

std::tuple<torch::Tensor, torch::Tensor> ComputeRelocationCUDA(
	torch::Tensor& opacity_old,
	torch::Tensor& scale_old,
	torch::Tensor& N,
	torch::Tensor& binoms,
	const int n_max)
{
	const int P = opacity_old.size(0);
  
	torch::Tensor final_opacity = torch::full({P}, 0, opacity_old.options().dtype(torch::kFloat32));
	torch::Tensor final_scale = torch::full({3 * P}, 0, scale_old.options().dtype(torch::kFloat32));

	if(P != 0)
	{
		UTILS::ComputeRelocation(P,
			opacity_old.contiguous().data<float>(),
			scale_old.contiguous().data<float>(),
			N.contiguous().data<int>(),
			binoms.contiguous().data<float>(),
			n_max,
			final_opacity.contiguous().data<float>(),
			final_scale.contiguous().data<float>());
	}

	return std::make_tuple(final_opacity, final_scale);

}

torch::Tensor MakeCategoryMaskCUDA(
	torch::Tensor& mask,
	int H, int W, int B
)
{
	auto int_ops = mask.options().dtype(torch::kInt32);
	torch::Tensor category_mask = torch::full({H, W}, 0.0, int_ops);


	UTILS::MakeCategoryMask(
		mask.contiguous().data<int>(),
		H, W, B,
		category_mask.contiguous().data<int>()
	);


	return category_mask;
}

torch::Tensor ExtractVisiblePointsCUDA(
	const torch::Tensor& orig_points,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	float boundary
)
{
	const int P = orig_points.size(0);
	const int B = viewmatrix.size(0);

	auto int_ops = orig_points.options().dtype(torch::kInt32);
	torch::Tensor visibility = torch::full({B, P}, 0.0, int_ops);

	UTILS::ExtractVisiblePoints(
		P, B, boundary,
		orig_points.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(), 
		visibility.contiguous().data<int>()
	);

	return visibility;
}